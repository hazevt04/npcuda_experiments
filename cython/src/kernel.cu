
#include <hip/hip_runtime.h>
#include <stdio.h>

void __global__ kernel_add_one(int* a, int length) {
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    if (gid < length) {
    	a[gid] += 1;
    }
}
