/*
This is the central piece of code. This file implements a class
(interface in gpuadder.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
using namespace std;

GPUAdder::GPUAdder (int* array_, int length_) {
  array = array_;
  length = length_;
  int size = length * sizeof(int);
  hipDeviceReset();
  hipDeviceProp_t prop;
  hipError_t err;
  err = hipGetDeviceProperties(&prop, 0);
  assert(prop.canMapHostMemory);
  err = hipSetDeviceFlags(hipDeviceMapHost);
  //cout << "Trying to hipSetDeviceFlags(hipDeviceMapHost)" << endl;
  assert(err == 0);
  err = hipHostRegister( (void*)array, size, hipHostRegisterMapped );
  //cout << "Trying to hipHostRegister( (void*)array, size, hipHostRegisterMapped )" << endl;
  assert(err == 0);
  err = hipHostGetDevicePointer( (void**)&array_d, (void*)array, 0);
  //cout << "Trying to hipHostGetDevicePointer( (void**)&array_d, (void*)array, 0)" << endl;

  // Prefetch array to GPU from CPU
  err = hipStreamAttachMemAsync(NULL, array_d, 0, hipMemAttachGlobal);
  hipStreamSynchronize(NULL);
  //cout << "End of constructor" << endl;
  //assert(err == 0);
}

void GPUAdder::increment() {
  kernel_add_one<<<64, 64>>>(array_d, length);
  //hipError_t err = hipGetLastError();
  //assert(err == 0);
}

void GPUAdder::retreive() {
  // Prefetch array to CPU from GPU
  hipError_t err = hipStreamAttachMemAsync(NULL, array_d, 0, hipMemAttachHost);
  hipStreamSynchronize(NULL); 
  //cout << "End of retreive()" << endl;
  //hipDeviceSynchronize();
}

void GPUAdder::retreive_to(int* array_, int length_) {
  assert(length == length_);
  int size = length * sizeof(int);
  hipError_t err = hipStreamAttachMemAsync(NULL, array_d, 0, hipMemAttachHost);
  hipStreamSynchronize(NULL); 
  memcpy( array_, array, size );
  //cout << "End of retreive_to()" << endl;
}

GPUAdder::~GPUAdder() {
}
